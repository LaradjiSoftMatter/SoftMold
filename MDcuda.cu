/**
 * \brief This simply runs a system through the timesteps. It can utilize OpenMP.
 * It is currently setup to use the implicit solvent molecular dynamic model used by Laradji's Computational Soft Matter lab. 
 * The particular system being studied is a phospholipid system. 
 * Various additions, such as cytoskeletons and nanoparticles, have been studied as well.
 * Multicomponent lipid systems have been studied.
 * This is the GPU (CUDA) version of the MD.cpp program.
**/

//These macros won't work on the GPU directly (as of Jun 14, 2022)
//Macros that can be overloaded in the library (comment them out if you don't want them):

//turning this flag off saves about 5% of runtime, but could also result in failure
//#define CELL_SIZE_FAILURE

//related to the above, if you turn on CELL_SIZE_FAILURE and receive a failure, make this bigger
//#define MAX_CELL_SIZE 8192

//sort optimization, testing, potentially better
//#define SORT_OPTIMIZATION

//omp molecules, testing, bullshit so far
//#define OMP_MOLECULES

//Enable error readout and halting
//#define ERRORS_ENABLED

//Enable warning readout and halting
//#define WARNINGS_ENABLED

//For anchor data
//#define ANCHOR_DATA

//only use the filled boxes
//#define LOW_DENSITY

//If the local density is high, the threads conflict, this fixes that by skipping some cells when low density is active
//#define LOW_DENSITY_SKIP 7

//a debugging flag
//#define FORCE_COUNT


//Include files from the library:

//For molecular dynamics forces and potentials
#include "include/MD.h"

//For the molecular dynamics variables
#include "include/system.h"

//For data extraction that is already enabled
//#include "dataExtraction.h"

//For GPU
#include "include/cuda/mpdCuda.h"

#include <ctime>

//#define VERLET_HALF_CARRY


//Macros that are used in this file:

//Flag to recenter the mass in the system
//#define RECENTER_MASS

//When to start computing diffusion, in tau
#define DIFFUSION_START 100.0

//frequency that the system is resized, 1=every time step, 2=every other time step, 3=every 3rd time step, etc...
#define resizeRate 8




int main(int argc, char* argv[])
{
	if(argc!=2)
	{
		//so simple, this can't possibly mess it up
		std::cerr << "usage: command name\n";
		return 0;
	}
	
	char *name=argv[1];
	std::string nameA(name);
	nameA+=".tmp";
	
	//the variables for the simulation, remember that for some reason constructor isn't explicit when nothing is in it
	Blob<float> System;
	
	//load variables, then initialize them, Script requires some functions from Blob
	Script<float, Blob <float> > fileIO(name,std::ios::in,&System);
	try {
		fileIO.read();
		fileIO.close();
	} catch(int e)
	{
		if(e==0)
		{
			fileIO.close();
			fileIO.open(nameA.c_str(),std::ios::in);
			fileIO.read();
			fileIO.close();
		}
		else
		{
			std::cerr << "Cannot open temp file! System load failure!" << std::endl;
			return -1;
		}
	}
	
	threeVector<float> *acc=System.getAccelerations();
	
	//for diffusion
	position <float> *aP=new position<float>[System.readNParticles()];
	
	//for xyz frames
	position <float> *p=System.getPositions();
	int nParticles=System.readNParticles();
	xyzFormat<float> xyzFile(p, nParticles);
	
	float trial=0, accepted=0;
	
	float resizeHistInterval=0.00001, maxInterval=System.readDeltaLXY();
	std::vector<float> resizeHist,rejectHist,dPotentialVal;
	if(System.readDeltaLXY()!=0)
	{
		//plus 1 for the end inclusion
		//resizeHist.alloc(static_cast<int>(System.readDeltaLXY()/resizeHistInterval)+1);
		int nIntervals=static_cast<int>(2.0*maxInterval/resizeHistInterval)+1;
		for(int i=0;i<nIntervals;i++)
			resizeHist.push_back(0);
		for(int i=0;i<nIntervals;i++)
			rejectHist.push_back(0);
	}
	
	
	//initialize the variables, objects, algorithms, data collection, etc...
	//for random size fluctuations
	std::mt19937 randGen(System.readSeed());
	threeVector<float> dL(System.readDeltaLXY(),System.readDeltaLXY(),0);
	mpd::barostat<float> bStat(dL, nParticles);
	mpd::randomAdaptor random(System.readSeed());
	mpd::interaction<float,3,2> bendList(nParticles);
	mpd::interaction<float,2,2> bondList(nParticles);
	//mpd::interaction<float,2,2> ballList(nParticles);
	std::vector<mpd::interaction<float,2,2>> ballLists;
	mpd::cell<float> cData(nParticles,System.readCutoff(),System.readSize(),System.readDeltaLXY());
	std::vector<float> mass(nParticles,1.0);
	
	//for data and resizing
	mpd::dataCollection<float> dataCollection(nParticles);
	
	//Add to the bond and bend lists
	for(int k=0;k<System.readNMolecules();k++)
	{
		auto mol=System.getMolecule()[k];
		switch(mol.readType())
		{
			case BOND:
			{
				auto b=mol.getBonds();
				auto c=mol.getConstants();
				for(int j=0;j<mol.readNBond();j++)
				{
					bondList.addInteraction(b[j].s[0],0,b[j].s,c);
					bondList.addInteraction(b[j].s[1],1,b[j].s,c);
				}
				break;
			}
			case BEND:
			{
				auto b=mol.getBonds();
				auto c=mol.getConstants();
				for(int j=0;j<mol.readNBond();j++)
				{
					bendList.addInteraction(b[j].s[0],0,b[j].s,c);
					bendList.addInteraction(b[j].s[1],1,b[j].s,c);
					bendList.addInteraction(b[j].s[2],2,b[j].s,c);
				}
				break;
			}
			case CHAIN:
			{
				auto b=mol.getBonds();
				auto c=mol.getConstants();
				for(int j=0;j<mol.readNBond();j++)
				{
					int start=b[j].s[START];
					int nChains=b[j].s[NCHAINS];
					int length=b[j].s[CHAINLENGTH];
					for(int i=start;i<start+length*nChains;i+=length)
					{
						std::vector<int> cha;
						for(int l=i;l<i+length;l++)
							cha.push_back(l);
						for(int l=0;l<length-2;l++)
						{
						bendList.addInteraction(cha[l],0,&(cha[l]),c+2);
						bendList.addInteraction(cha[l+1],1,&(cha[l]),c+2);
						bendList.addInteraction(cha[l+2],2,&(cha[l]),c+2);
						}
						for(int l=0;l<length-1;l++)
						{
						bondList.addInteraction(cha[l],0,&(cha[l]),c);
						bondList.addInteraction(cha[l+1],1,&(cha[l]),c);
						}
					}
				}
				
				break;
			}
			case BALL:
			{
				mpd::interaction<float,2,2> ballList(nParticles);
				auto b=mol.getBonds();
				auto c=mol.getConstants();
				mass[b[0].s[0]]=(4.0)*M_PI*c[0]*c[0];
				for(int j=0;j<mol.readNBond();j++)
				{
					//ballList.addInteraction(b[j].s[0],0,b[j].s,c);
					ballList.addInteraction(b[j].s[1],1,b[j].s,c);
				}
				ballLists.emplace_back(ballList);
				break;
			}
			default:
			{
				std::cerr << "Molecule type " << mol.readType() 
					<< " not yet implemented!" << std::endl;
				break;
			}
		}
	}
	
	mpd::state<float> state(System.getPositions(),System.getVelocities(),System.getAccelerations(),
				System.getTwoBodyFconst(),System.getTwoBodyUconst(),
				nParticles,System.readNTypes(),System.readDeltaT(),System.readGamma(),
				System.readInitialTemp(),System.readSize(),mass.data());
	
	//Other initializations
	std::string framesFilename("frames_");
	framesFilename+=name;
	framesFilename+=".xyz";
	
	std::string potentialFileName("potential_");
	potentialFileName+=name;
	potentialFileName+=".dat";
	
	std::string kineticFileName("kinetic_");
	kineticFileName+=name;
	kineticFileName+=".dat";
	
	std::string sizeFileName("size_");
	sizeFileName+=name;
	sizeFileName+=".dat";
	
	//Send to device
	bondList.toDevice();
	for(auto &ballList:ballLists)
		ballList.toDevice();
	bendList.toDevice();
	state.toDevice();
	
	//molecular dynamics forces
	mpd::zeroRange_device(state.deviceState().a,nParticles);
	mpd::bondForces_device(bondList.deviceInteraction(),state.deviceState());
	for(auto &ballList:ballLists)
		mpd::ballForces_device(ballList.deviceInteraction(),state.deviceState());
	mpd::bendForces_device(bendList.deviceInteraction(),state.deviceState());
	mpd::cellComputeForce_device(cData.deviceCell(), state.deviceState());
	mpd::applyMass_device(state.deviceState());
	
	
	//this corrects an issue where an extra data point is added when the system is restarted
	if(System.readInitialTime()==0)
	{
		mpd::zeroRange_device(dataCollection.deviceState().kineticEnergy,nParticles);
		mpd::kinetic_device(state.deviceState(),dataCollection.deviceState());
		float kinetic=
			mpd::reduceRange_device(dataCollection.deviceState().kineticEnergy,nParticles);
		
		mpd::zeroRange_device(dataCollection.deviceState().potentialEnergy,nParticles);
		mpd::bondPotential_device(bondList.deviceInteraction(),state.deviceState(),
					  dataCollection.deviceState());
		for(auto &ballList:ballLists)
			mpd::ballPotential_device(ballList.deviceInteraction(),state.deviceState(),
					  dataCollection.deviceState());
		mpd::bendPotential_device(bendList.deviceInteraction(),state.deviceState(),
					  dataCollection.deviceState());
		mpd::cellComputePotential_device(cData.deviceCell(),state.deviceState(),
					  dataCollection.deviceState());
		float potential=
			mpd::reduceRange_device(dataCollection.deviceState().potentialEnergy,nParticles);
		
		std::fstream potentialFile(potentialFileName, std::ios::out | std::ios::app);
		potentialFile << "0\t" << potential << std::endl;
		
		std::fstream kineticFile(kineticFileName, std::ios::out | std::ios::app);
		kineticFile << "0\t" << kinetic << std::endl;
		
		xyzFile.open(framesFilename,std::ios::out | std::ios::app);
		xyzFile.store();
		xyzFile.close();
	}
	else
	{
		//Surprise! This is done because a previously run configuration don't do this upon exit
		mpd::verletSecond_device(state.deviceState());
	}
	//using integer indexing, the 0.0000001 fixes an accuracy issue with the gnu c++ compiler.
	//Don't believe it affects anything else...
	int endInt=int(System.readFinalTime()/System.readDeltaT()+0.0000001);//end
	int startInt=int(System.readInitialTime()/System.readDeltaT()+0.0000001);//start
	
	int storeint=int(System.readStoreInterval()/System.readDeltaT()+0.0000001);//when to store
	int measureint=int(System.readMeasureInterval()/System.readDeltaT()+0.0000001);//when to measure
	
	
	int tempStepInt=0;
	float tempStep=0;
	if(System.readTempStepInterval()>0)
	{
		tempStep=System.readTempStepInterval()*(System.readFinalTemp()-System.readInitialTemp())/
			 (System.readFinalTime()-System.readInitialTime());
		tempStepInt=(endInt-startInt)/int((System.readFinalTime()-System.readInitialTime())/
			    (System.readTempStepInterval()));
	}
	
	std::cerr << "starting main loop: \n";
	
	time_t current=time(NULL);
	
	bool exitFlag=false;//for premature exits!
	
	//the molecular dynamics loop, the "running" of the system
	for(int i=startInt;i<=endInt && !exitFlag;i++)
	{
		System.setInitialTime((float)i*System.readDeltaT());
		
		mpd::verletFirst_device(state.deviceState());
		mpd::zeroRange_device(state.deviceState().a,nParticles);
		//The system is stored here because force is updated here, but velocities are updated next.
		//It causes a problem when it reenters the loop from a previously run configuration.
		if(System.readTempStepInterval()>0)
			if(i%tempStepInt==0 && tempStepInt!=0 && i<endInt)
				System.setInitialTemp(System.readInitialTemp()+tempStep);
		
		if(i%storeint==0 && i!=startInt)
		{
			state.toHost();
			fileIO.open(nameA.c_str(),std::ios::out);
			fileIO.write();
			fileIO.close();
			
			fileIO.open(name,std::ios::out);
			fileIO.write();
			fileIO.close();
			
			xyzFile.open(framesFilename,std::ios::out | std::ios::app);
			xyzFile.store();
			xyzFile.close();
		}
		state.temperature=System.readInitialTemp();
		mpd::langevin_device(state.deviceState(),random.deviceState());
		mpd::cellComputeForce_device(cData.deviceCell(), state.deviceState());
		mpd::bondForces_device(bondList.deviceInteraction(),state.deviceState());
		for(auto &ballList:ballLists)
			mpd::ballForces_device(ballList.deviceInteraction(),state.deviceState());
		mpd::bendForces_device(bendList.deviceInteraction(),state.deviceState());
		mpd::applyMass_device(state.deviceState());
		mpd::verletSecond_device(state.deviceState());
		
		//Measurements are output here
		if(i%measureint==0 && i!=startInt)
		{
			//float last=current;
			time_t last=current;
			current=time(NULL);
			//time since last storage step, good for benchmarking
			std::cerr << System.readInitialTime() << '\t' << current-last << std::endl;
			
			//Data calculations that we are interested in starting
			
			//Data calculations
			mpd::zeroRange_device(dataCollection.deviceState().kineticEnergy,nParticles);
			mpd::kinetic_device(state.deviceState(),dataCollection.deviceState());
			float kinetic=
				mpd::reduceRange_device(dataCollection.deviceState().kineticEnergy,nParticles);
			
			mpd::zeroRange_device(dataCollection.deviceState().potentialEnergy,nParticles);
			mpd::bondPotential_device(bondList.deviceInteraction(),state.deviceState(),
						  dataCollection.deviceState());
			for(auto &ballList:ballLists)
				mpd::ballPotential_device(ballList.deviceInteraction(),state.deviceState(),
						  dataCollection.deviceState());
			mpd::bendPotential_device(bendList.deviceInteraction(),state.deviceState(),
						  dataCollection.deviceState());
			mpd::cellComputePotential_device(cData.deviceCell(),state.deviceState(),
						  dataCollection.deviceState());
			float potential=
				mpd::reduceRange_device(dataCollection.deviceState().potentialEnergy,nParticles);
			
			//dataCollection.toHost();
			//state.toHost();
			
			std::fstream potentialFile(potentialFileName, std::ios::out | std::ios::app);
			potentialFile << System.readInitialTime() << "\t" << potential << std::endl;
			
			std::fstream kineticFile(kineticFileName, std::ios::out | std::ios::app);
			kineticFile << System.readInitialTime() << "\t" << kinetic << std::endl;
			
			threeVector<float> size=System.readSize();
			std::fstream sizeFile(sizeFileName, std::ios::out | std::ios::app);
			sizeFile << System.readInitialTime() << '\t' << size.x << '\t' << size.y << '\t' << size.z << std::endl;
				
			current=time(NULL);
		}
		//short section to resize system, note that it only works when deltaLXY 
		// is something other than 0, it flags execution.
		if(i%resizeRate==0 && i!=0 && System.readDeltaLXY()!=0)
		{
			threeVector<float> oldSize=System.readSize();
			//threeVector<float> fluctuation=bStat.dependentFluctuationConstantVolume(randGen, oldSize);
			threeVector<float> fluctuation=bStat.independentFluctuationConstantVolume(randGen, oldSize);
			
			threeVector<float> newSize=oldSize;
			newSize.x+=fluctuation.x;
			newSize.y+=fluctuation.y;
			newSize.z+=fluctuation.z;
			threeVector<float> scale=newSize;
			scale.x/=oldSize.x;
			scale.y/=oldSize.y;
			scale.z/=oldSize.z;
			
			mpd::zeroRange_device(bStat.deviceState().dPotential,nParticles);
			mpd::bondDPotential_device(bondList.deviceInteraction(),state.deviceState(),
						  bStat.deviceState(),scale);
			for(auto &ballList:ballLists)
				mpd::ballDPotential_device(ballList.deviceInteraction(),state.deviceState(),
						  bStat.deviceState(),scale);
			mpd::bendDPotential_device(bendList.deviceInteraction(),state.deviceState(),
						  bStat.deviceState(),scale);
			mpd::cellComputeDPotential_device(cData.deviceCell(),state.deviceState(),
						  bStat.deviceState(),scale);
			float dPotential=
				mpd::reduceRange_device(bStat.deviceState().dPotential,nParticles);
			if(bStat.MCtest(dPotential, System.readInitialTemp(), 0.0, 
				System.readTension(),oldSize,newSize, randGen))
			{
				//this can't perform the state resize as of yet, but can rescale the 
				//device positions
				mpd::rescale_device(state.deviceState(),scale);
				//this should be the only resize for everything
				state.resize(newSize);
				//this could be deleted if the above is universal
				cData.resize(newSize,System.readDeltaLXY());
				System.setSize(newSize);
				resizeHist[(fluctuation.x+maxInterval)/resizeHistInterval]+=0.5;
				resizeHist[(fluctuation.y+maxInterval)/resizeHistInterval]+=0.5;
				//int rIndex=(fluctuation.x+System.readDeltaLXY())/resizeHistInterval;
				
				accepted++;
			}
			else
			{
				rejectHist[(fluctuation.x+maxInterval)/resizeHistInterval]+=0.5;
				rejectHist[(fluctuation.y+maxInterval)/resizeHistInterval]+=0.5;
			}
			trial++;
		}
	}
	
	if(System.readDeltaLXY()!=0)
	{	
		std::fstream resizeHistFile;
		std::string resizeHistFileName="resizeHist_";
		resizeHistFileName+=argv[1];
		resizeHistFileName+=".dat";
		
		resizeHistFile.open(resizeHistFileName.c_str(),std::ios::out);
		if(resizeHistFile.is_open())
		{
			for(int i=0;i<resizeHist.size();i++)
				resizeHistFile << (static_cast<float>(i)*resizeHistInterval)-maxInterval\
					 << '\t' << resizeHist[i] << '\t' << rejectHist[i] << std::endl;
			resizeHistFile.close();
		}
		std::cerr << "Resize acceptance ratio: " << accepted/trial << std::endl;
	}
	
	return 0;
}

